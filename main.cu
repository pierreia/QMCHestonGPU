#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <time.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
//#include "dev_array.h"
#include <hiprand.h>
#include <random>
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "product.h"

using namespace std;

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)



#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)


double normal_random()
{
    static std::default_random_engine e(1234);
    static std::normal_distribution<> normal;
    return normal(e);
}

int main() {
    try {
        // declare variables and constants
        int N_PATHS = 10000;
        int N_STEPS = 1000;
        

        const float kappa = 6.21;
        const float theta = 0.019;
        const float sigma = 0.61;
        const float v0 = 0.010201;
        const float T = 1;
        const float r = 0.0319;
        const float s0 = 100;
        const float K = 100;
        const float rho = -0.7;
        float dt = T/float(N_STEPS);





        float gpu_price;
        
        double t1=double(clock())/CLOCKS_PER_SEC;
        //MCEuro(kappa, theta, sigma, v0, T, r, s0, K, rho, N_STEPS, N_PATHS, &gpu_price);

        OptionPriceResult option_result = {
            EURO,
            MILSTEIN,
            PSEUDO,
            kappa,
            theta,
            sigma,
            v0,
            T,
            r,
            s0,
            K,
            rho,
            N_STEPS,
            N_PATHS,
            0., // price will be set by calculateOptionPrice
            0.0 // execution_time will be set by calculateOptionPrice
        };

        calculateOptionPrice(option_result);
        
        double t2=double(clock())/CLOCKS_PER_SEC;

        float *d_S;
        float *h_S;
        checkCudaErrors(hipMalloc((void **)&d_S, sizeof(float) * N_PATHS));

        
        h_S = (float*)malloc(sizeof(float) * N_PATHS);

        /* Generation with Curand State */

        hiprandState *devStates;
        hiprandStateMRG32k3a *devMRGStates;
        CUDA_CALL(hipMalloc((void **)&devStates, N_PATHS *
                  sizeof(hiprandState)));

        CUDA_CALL(hipMalloc((void **)&devMRGStates, N_PATHS *
                  sizeof(hiprandStateMRG32k3a)));
        

        //set value to zero
        CUDA_CALL(hipMemset(d_S, 0, N_PATHS *
              sizeof(unsigned int)));

        const unsigned BLOCK_SIZE = 512;
        const unsigned GRID_SIZE = ceil(float(N_PATHS) / float(BLOCK_SIZE));
        setup_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(devMRGStates);
        heston_kernel_curand<<<GRID_SIZE, BLOCK_SIZE>>>(devMRGStates, kappa, theta, sigma, v0, T, r, s0, K, rho, N_STEPS, N_PATHS, d_S);
        checkCudaErrors(hipMemcpy(h_S, d_S, sizeof(float) * N_PATHS, hipMemcpyDeviceToHost));
        

        // compute the payoff average
        double temp_sum2=0.0;
        for(size_t i=0; i<N_PATHS; i++) {
            temp_sum2 +=h_S[i];
        }
        
        double gpu_price2 = temp_sum2/N_PATHS;

        //Set values to zero
        checkCudaErrors(hipMemset(d_S, 0,
                            N_PATHS * sizeof(float)));
        
        memset(h_S, 0, N_PATHS * sizeof(float));

        double t5=double(clock())/CLOCKS_PER_SEC;

        /* END OF GENERATION WITH CURAND PSEUDORANDOM */

        /* START OF GENERATION WITH CURAND QUASIRANDOM */
        hiprandStateScrambledSobol32 *devSobol32States;
        hiprandDirectionVectors32_t *hostVectors32;
        unsigned int * hostScrambleConstants32;
        unsigned int * devDirectionVectors32;
        unsigned int * devScrambleConstants32;
        const int VECTOR_SIZE = 32; 


        /* Set results to 0 */
        checkCudaErrors(hipMemset(d_S, 0,
                            N_PATHS * sizeof(float)));
        
        memset(h_S, 0, N_PATHS * sizeof(float));
        

        /* Get pointers to the 32 bit scrambled direction vectors and constants*/
        CURAND_CALL(hiprandGetDirectionVectors32( &hostVectors32,
                                                HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6));

        CURAND_CALL(hiprandGetScrambleConstants32( &hostScrambleConstants32));


        /* Allocate memory for 3 states per thread (x, y, z), each state to get a unique dimension */
        checkCudaErrors(hipMalloc((void **)&devSobol32States,
                N_PATHS * 2 * sizeof(hiprandStateScrambledSobol32)));

        /* Allocate memory and copy 3 sets of vectors per thread to the device */

        checkCudaErrors(hipMalloc((void **)&(devDirectionVectors32),
                            N_PATHS * 2 * VECTOR_SIZE * sizeof(long long int)));

        checkCudaErrors(hipMemcpy(devDirectionVectors32, hostVectors32,
                            N_PATHS * 2 * VECTOR_SIZE * sizeof(long long int),
                            hipMemcpyHostToDevice));

        /* Allocate memory and copy 3 scramble constants (one costant per dimension)
        per thread to the device */

        checkCudaErrors(hipMalloc((void **)&(devScrambleConstants32),
                            N_PATHS * 2 * sizeof(long long int)));

        checkCudaErrors(hipMemcpy(devScrambleConstants32, hostScrambleConstants32,
                            N_PATHS * 2 * sizeof(long long int),
                            hipMemcpyHostToDevice)); 

        /* Initialize the states */

        setup_kernel<<<BLOCK_SIZE, GRID_SIZE>>>(devDirectionVectors32,
                                                        devScrambleConstants32,
                                                        devSobol32States);

        /* Generate and count quasi-random points  */

        
        heston_kernel_curand<<<BLOCK_SIZE, GRID_SIZE>>>(devSobol32States, kappa, theta, sigma, v0, T, r, s0, K, rho, N_STEPS, N_PATHS, d_S);
        checkCudaErrors(hipMemcpy(h_S, d_S, sizeof(float) * N_PATHS, hipMemcpyDeviceToHost));
        
        // compute the payoff average
        double temp_sum_qmc=0.0;
        for(size_t i=0; i<N_PATHS; i++) {
            temp_sum_qmc +=h_S[i];
        }
        
        double gpu_price_qmc = temp_sum_qmc/N_PATHS;
        double t6=double(clock())/CLOCKS_PER_SEC;

        float sum_price = 0;

        float h_z_m = 0.0;
        int idx;

        for (int i = 0; i < N_PATHS; ++i) {
            float v = v0;
            float s = s0;
            
            idx = i*N_STEPS;

            //float v_plus, s_plus;

            for (int j = 0; j < N_STEPS; ++j) {

                //float z1 = h_Z[2*idx];
                //float z2 = h_Z[2*idx + 1];


                float z1 = normal_random();
                float z2 = normal_random();

                h_z_m += z1+z2;

                float dw1 = z1;
                float dw2 = rho * z1 + sqrt(1 - rho * rho) * z2;

                //float v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * dw2;
                //float s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * dw1);

                float v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * sqrt(dt) * dw2 + 0.25*sigma*sigma*dt*(dw2*dw2 - 1); // Milstein
                float s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1);

                //float s_plus = s*(1 + r*dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1 + s*0.25*dt*(dw1*dw1 - 1));

                v = max(v_plus, 0.0);
                s = max(s_plus, 0.0);

                idx++;
            }

            float payoff = max(s - K, 0.0);
            sum_price += exp(-r * T) * payoff;
            
        }

        h_z_m/= idx;

        double cpu_price = sum_price/N_PATHS;

        double t7=double(clock())/CLOCKS_PER_SEC;



        double t8=double(clock())/CLOCKS_PER_SEC;

        

        /* Asian Generation with Curand State */

        int m = 4;
        //hiprandState *devStates;
        //hiprandStateMRG32k3a *devMRGStates;
        //CUDA_CALL(hipMalloc((void **)&devStates, N_PATHS *
        //          sizeof(hiprandState)));

        //CUDA_CALL(hipMalloc((void **)&devMRGStates, N_PATHS *
        //          sizeof(hiprandStateMRG32k3a)));
        

        //set value to zero
        CUDA_CALL(hipMemset(d_S, 0, N_PATHS *
              sizeof(unsigned int)));
        
        float *h_delta;
        float *d_delta;

        checkCudaErrors(hipMalloc((void **)&d_delta, sizeof(float) * N_PATHS));

        h_delta = (float*)malloc(sizeof(float) * N_PATHS);
        
        setup_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(devMRGStates);
        heston_kernel_asian<<<GRID_SIZE, BLOCK_SIZE>>>(devMRGStates, kappa, theta, sigma, v0, T, r, s0, K, rho, N_STEPS, N_PATHS, d_S, d_delta, m);
        
        checkCudaErrors(hipMemcpy(h_S, d_S, sizeof(float) * N_PATHS, hipMemcpyDeviceToHost));
        
        checkCudaErrors(hipMemcpy(h_delta, d_delta, sizeof(float) * N_PATHS, hipMemcpyDeviceToHost));
        
        // compute the payoff average
        double temp_asian=0.0;
        double temp_delta=0.0;
        for(size_t i=0; i<N_PATHS; i++) {
            temp_asian +=h_S[i];
            temp_delta +=h_delta[i];
        }
        
        double gpu_asian = temp_asian/N_PATHS;
        double asian_delta = temp_delta/N_PATHS;
        double t9=double(clock())/CLOCKS_PER_SEC;

        cout<<"****************** INFO ******************\n";
        cout<<"Number of Paths: " << N_PATHS << "\n";
        cout<<"Underlying Initial Price: " << s0 << "\n";
        cout<<"Initial Variance: " << v0 << "\n";
        cout<<"Strike: " << K << "\n";
        cout<<"Time to Maturity: " << T << " years\n";
        cout<<"Risk-free Interest Rate: " << r << "%\n";
        cout<<"Annual drift: " << theta << "%\n";
        cout<<"Volatility: " << sigma << "%\n";
        cout<<"****************** PRICE ******************\n";
        cout<<"Option Price (GPU): " << option_result.price << "\n";
        cout<<"Option Price MC (GPU): " << gpu_price2 << "\n";
        cout<<"Option Price QMC (GPU): " << gpu_price_qmc << "\n";
        cout<<"Option Price (CPU): " << cpu_price << "\n";
        cout<<"Option Price (Real): 6.8061 \n";
        cout<<"Asian Option Price (GPU): " << gpu_asian << "\n";
        cout<<"Asian Option Delta (GPU): " << asian_delta << "\n";
        cout<<"******************* TIME *****************\n";
        cout<<"GPU Monte Carlo Computation: " << option_result.execution_time*1e3 << " ms\n";
        cout<<"GPU Quasi Monte Carlo Computation: " << (t6-t5)*1e3 << " ms\n";
        cout<<"CPU Monte Carlo Computation: " << (t7-t6)*1e3 << " ms\n";
        cout<<"Speed up Factor: " << (t7-t6)/(t2-t1) << "\n";
        
        cout<<"h_Z mean:" << h_z_m << "\n";
        cout<<"******************* END *****************\n";
        // destroy generator
        //hiprandDestroyGenerator( curandGenerator ) ;

    /* Cleanup */

        checkCudaErrors(hipFree(devSobol32States));
        checkCudaErrors(hipFree(devDirectionVectors32));
        checkCudaErrors(hipFree(devScrambleConstants32));
        checkCudaErrors(hipFree(d_S));
        free(h_S);
    }
    catch(exception& e) {
        cout<< "exception: " << e.what() << "\n";
    }
}