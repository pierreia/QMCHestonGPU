#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandStateMRG32k3a *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(12345, id, 0, &state[id]);
}


/* This kernel initializes state per thread for each of x, y, and z */

__global__ void setup_kernel(unsigned int * sobolDirectionVectors,
                             unsigned int *sobolScrambleConstants,
                             hiprandStateScrambledSobol32 *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int dim = 2*id;
    int const VECTOR_SIZE = 32;
    /* Each thread uses 3 different dimensions */
    hiprand_init(sobolDirectionVectors + VECTOR_SIZE*dim,
                sobolScrambleConstants[dim],
                1234,
                &state[dim]);

    hiprand_init(sobolDirectionVectors + VECTOR_SIZE*(dim + 1),
                sobolScrambleConstants[dim + 1],
                1234,
                &state[dim + 1]);

}

__global__ void heston_kernel_curand(hiprandStateMRG32k3a *state, float kappa, float theta, float sigma, float v0, float T, float r, float s0, float K, float rho, int N_timesteps, int N_paths, float *d_S)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    //unsigned int count = 0;
    float2 x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random normals */

    float k_payoff = 0.;

    float v = v0;
    float s = s0;

    float dt = T/N_timesteps;

    float v_plus, s_plus;

    
    if (id < N_paths) {

            for (int j = 0; j < N_timesteps; ++j) {
                x = hiprand_normal2(&localState);
                float z1 = x.x;
                float z2 = x.y;

                float dw1 = z1;
                float dw2 = rho * z1 + sqrt(1 - rho * rho) * z2;

                //float v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * dw2; //Euler
                //float s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * dw1);


                v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * sqrt(dt) * dw2 + 0.25*sigma*sigma*dt*(dw2*dw2 - 1); // Milstein
                s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1);

                //float s_plus = s*(1 + r*dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1 + s*0.25*dt*(dw1*dw1 - 1));

                v = max(v_plus, 0.0);
                s = max(s_plus, 0.0);

                
            
            }
        float payoff = max(s - K, 0.0);
        k_payoff += payoff;
        d_S[id] = exp(-r * T) * k_payoff;
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
}


__global__ void heston_kernel_curand(hiprandStateScrambledSobol32 *state, float kappa, float theta, float sigma, float v0, float T, float r, float s0, float K, float rho, int N_timesteps, int N_paths, float *d_S)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int baseDim = 2 * id;
    //unsigned int count = 0;
    //float2 x;
    /* Copy state to local memory for efficiency */
    //hiprandStateScrambledSobol32 localState = state[id];
    /* Generate pseudo-random normals */

    

    float v = v0;
    float s = s0;

    float dt = T/N_timesteps;

    float v_plus, s_plus;
    
    float z1, z2;
    
    if (id < N_paths) {
            for (int j = 0; j < N_timesteps; ++j) {
                z1 = hiprand_normal(&state[baseDim]);
                z2 = hiprand_normal(&state[baseDim + 1]);
                
                float dw1 = z1;
                float dw2 = rho * z1 + sqrt(1 - rho * rho) * z2;

                //float v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * dw2; //Euler
                //float s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * dw1);


                v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * sqrt(dt) * dw2 + 0.25*sigma*sigma*dt*(dw2*dw2 - 1); // Milstein
                s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1);

                //s_plus = s + s*r*dt + s * sqrt(v*dt)* dw1 + s*s*0.25*dt*(dw1*dw1 - 1);

                v = max(v_plus, 0.0);
                s = max(s_plus, 0.0);

                
            
            }
        float payoff = max(s - K, 0.0);
        d_S[id] = exp(-r * T) * payoff;
    }
    /* Copy state back to global memory */
    
    /* Store results */
}

__global__ void heston_kernel(float kappa, float theta, float sigma, float v0, float T, float r, float s0, float K, float rho, int N_timesteps, int N_paths, float *d_S, float * d_Z){
        const unsigned tid = threadIdx.x;
        const unsigned bid = blockIdx.x;
        const unsigned bsz = blockDim.x;
        int s_idx = tid + bid * bsz;
        int n_idx = tid + bid * bsz;

        float k_payoff = 0.;

        float v = v0;
        float s = s0;

        float dt = T/N_timesteps;

        float v_plus, s_plus;

        
        if (s_idx < N_paths) {

                for (int j = 0; j < N_timesteps; ++j) {
                    float z1 = d_Z[2*n_idx];
                    float z2 = d_Z[2*n_idx+1];

                    float dw1 = z1;
                    float dw2 = rho * z1 + sqrt(1 - rho * rho) * z2;

                    //float v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * dw2; //Euler
                    //float s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * dw1);


                    v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * sqrt(dt) * dw2 + 0.25*sigma*sigma*dt*(dw2*dw2 - 1); // Milstein
                    s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1);

                    //float s_plus = s*(1 + r*dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1 + s*0.25*dt*(dw1*dw1 - 1));

                    v = max(v_plus, 0.0);
                    s = max(s_plus, 0.0);

                    n_idx ++;
                
                }
            float payoff = max(s - K, 0.0);
            d_S[s_idx] = exp(-r * T) * payoff;
        }
}

void heston_euro_call(
        float kappa, float theta, float sigma, float v0, float T, float r, float s0, float K, float rho, int N_timesteps, int N_paths, float *d_S, float * d_Z) {
        const unsigned BLOCK_SIZE = 512;
        const unsigned GRID_SIZE = ceil(float(N_paths) / float(BLOCK_SIZE));
        heston_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(
        kappa, theta, sigma, v0, T, r, s0, K, rho, N_timesteps, N_paths, d_S, d_Z);
    }



__global__ void heston_kernel_asian(hiprandStateMRG32k3a *state, float kappa, float theta, float sigma, float v0, float T, float r, float s0, float K, float rho, int N_timesteps, int N_paths, float *d_S, float *d_delta, int m)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    //unsigned int count = 0;
    float2 x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random normals */



    float v = v0;
    float s = s0;
    float s_mean = 0.;
    float delta = 0.;

    float dt = T/N_timesteps;

    float v_plus, s_plus;
    
    
    
    if (id < N_paths) {

            for (int j = 0; j < N_timesteps; ++j) {
                x = hiprand_normal2(&localState);
                float z1 = x.x;
                float z2 = x.y;

                float dw1 = z1;
                float dw2 = rho * z1 + sqrt(1 - rho * rho) * z2;

                //float v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * dw2; //Euler
                //float s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * dw1);


                v_plus = v + kappa * (theta - max(v, 0.0)) * dt + sigma * sqrt(max(v, 0.0)) * sqrt(dt) * dw2 + 0.25*sigma*sigma*dt*(dw2*dw2 - 1); // Milstein
                s_plus = s * exp((r - 0.5 * max(v, 0.0)) * dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1);

                //float s_plus = s*(1 + r*dt + sqrt(max(v, 0.0)) * sqrt(dt)* dw1 + s*0.25*dt*(dw1*dw1 - 1));

                v = max(v_plus, 0.0);
                s = max(s_plus, 0.0);

                if (j == 249) {
                    s_mean += s;
                } else if (j == 449) {
                    s_mean += s;
                } else if (j == 749) {
                    s_mean += s;
                } else if (j == 999) {
                    s_mean += s;
                }
                
            
            }
        s_mean/=m;
        float payoff = max(s_mean - K, 0.0);

        d_S[id] = exp(-r * T) * payoff;

        if (s_mean > K) {
            delta = exp(-r * T) * s_mean/s0;
        }
        d_delta[id] = delta;
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
}